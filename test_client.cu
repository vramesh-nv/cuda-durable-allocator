#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <fcntl.h>
#include <errno.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <sys/wait.h>
#include <sys/xattr.h>
#include <getopt.h>
#include <assert.h>

#define CUDA_CHECK(err) do { \
    hipError_t _err = (err); \
    if (_err != hipSuccess) { \
        printf("CUDA error %d: %s\n", _err, hipGetErrorString(_err)); \
        return -1; \
    } \
} while (0)

#define CUDA_CHECK_DRV(err) do { \
    hipError_t _err = (err); \
    if (_err != hipSuccess) { \
        const char* error_str; \
        hipDrvGetErrorString(_err, &error_str); \
        printf("CUDA error %d: %s\n", _err, error_str); \
        return -1; \
    } \
} while (0)

// Test client for the GPU Memory FUSE filesystem
// Tests the simplified create + truncate workflow

#define TEST_MOUNT_PATH "./test_mount"

void print_test_header(const char *test_name) {
    printf("\n=== %s ===\n", test_name);
}

void print_error(const char *operation) {
    printf("ERROR in %s: %s\n", operation, strerror(errno));
}

void print_usage(const char *program_name) {
    printf("Usage: %s [OPTIONS]\n", program_name);
    printf("Options:\n");
    printf("  --parent    Run as parent process (creates allocation and waits for child)\n");
    printf("  --child     Run as child process (accesses existing allocation)\n");
    printf("  --help      Show this help message\n");
    printf("\nExample:\n");
    printf("  # Terminal 1 (parent):\n");
    printf("  %s --parent\n", program_name);
    printf("  \n");
    printf("  # Terminal 2 (child):\n");
    printf("  %s --child\n", program_name);
}

__global__ void kernel_write(void *ptr, size_t size) {
    for (size_t i = 0; i < size; i++) {
        ((char *)ptr)[i] = (unsigned char)i;
    }
}

__global__ void kernel_read(void *ptr, size_t size) {
    for (size_t i = 0; i < size; i++) {
        unsigned char *ptr_char = (unsigned char *)ptr;
        //printf("%d, %zd\n", (int)ptr_char[i], i);
        assert(ptr_char[i] == (unsigned char)i);
    }
}

static hipDeviceptr_t
get_va_from_fabric_handle(CUmemFabricHandle fabric_handle, size_t allocation_size, size_t granularity) {
    hipMemGenericAllocationHandle_t gpu_handle;
    CUDA_CHECK_DRV(hipMemImportFromShareableHandle(&gpu_handle, (void *)&fabric_handle, CU_MEM_HANDLE_TYPE_FABRIC));
    
    hipDeviceptr_t va = 0U;
    CUDA_CHECK_DRV(hipMemAddressReserve(&va, allocation_size, granularity, 0U, 0));

    CUDA_CHECK_DRV(hipMemMap(va, allocation_size, 0, gpu_handle, 0));
    CUDA_CHECK_DRV(hipMemRelease(gpu_handle));

    hipMemAccessDesc accessDesc;
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = 0;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    CUDA_CHECK_DRV(hipMemSetAccess(va, allocation_size, &accessDesc, 1));

    return va;
}

int test_parent_process() {
    print_test_header("PARENT PROCESS - Creating GPU Allocation");
    
    char path[256];
    snprintf(path, sizeof(path), "%s/shared_gpu_buffer", TEST_MOUNT_PATH);
    
    // 1. Create the file (no GPU memory allocated yet)
    printf("1. Creating file (no GPU memory yet)...\n");
    mode_t mode = S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH; 
    int fd = creat(path, mode);
    if (fd < 0) {
        print_error("open/create");
        return -1;
    }
    close(fd);  // Close the creation file descriptor

    printf("2. Truncating to 8MB to allocate GPU memory...\n");
    if (truncate(path, 8 * 1024 * 1024) != 0) {
        print_error("truncate");
        return -1;
    }

    // 3. Get allocation size using getxattr
    printf("3. Getting allocation size using getxattr...\n");
    char size_str[64];
    ssize_t size_len = getxattr(path, "user.allocation_size", size_str, sizeof(size_str) - 1);
    if (size_len < 0) {
        print_error("getxattr allocation_size");
        return -1;
    }
    size_str[size_len] = '\0';
    size_t allocation_size = atol(size_str);
    printf("   Retrieved allocation size: %s bytes (%.2f MB)\n", 
           size_str, allocation_size / (1024.0 * 1024.0));

    // 4. List all available extended attributes
    printf("4. Listing available extended attributes...\n");
    char attr_list[1024];
    ssize_t list_size = listxattr(path, attr_list, sizeof(attr_list));
    if (list_size < 0) {
        print_error("listxattr");
    } else {
        printf("   Available attributes (%zd bytes):\n", list_size);
        char *attr = attr_list;
        while (attr < attr_list + list_size) {
            printf("   - %s\n", attr);
            attr += strlen(attr) + 1;
        }
    }

    CUmemFabricHandle fabric_handle;
    ssize_t bytes_read = getxattr(path, "user.fabric_handle", &fabric_handle, sizeof(CUmemFabricHandle));
    if (bytes_read != sizeof(CUmemFabricHandle)) {
        printf("getxattr failed: expected %zu bytes, got %zd bytes\n", sizeof(CUmemFabricHandle), bytes_read);
        print_error("getxattr");
        return -1;
    }
    
    printf("4. Successfully read fabric handle (%zu bytes)\n", sizeof(CUmemFabricHandle));
    
    // 5. Initialize CUDA
    CUDA_CHECK_DRV(hipInit(0));
    
    hipDeviceptr_t va = get_va_from_fabric_handle(fabric_handle, allocation_size, allocation_size);

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    kernel_write<<<1, 1, 0, stream>>>((void *)va, allocation_size);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    
    // Wait for user input (simulating child process completion)
    //getchar();
    
    printf("5. Parent process completed.\n");
    printf("✅ PARENT PROCESS completed successfully!\n");
    return 0;
}

int test_child_process() {
    print_test_header("CHILD PROCESS - Accessing Shared GPU Allocation");
    
    char path[256];
    snprintf(path, sizeof(path), "%s/shared_gpu_buffer", TEST_MOUNT_PATH);
    
    // 1. Check if the allocation exists
    printf("1. Checking if shared allocation exists...\n");
    struct stat st;
    if (stat(path, &st) != 0) {
        print_error("stat - allocation not found");
        printf("   Make sure to run the parent process first!\n");
        return -1;
    }
    printf("   Found allocation: %ld bytes (%.2f MB)\n", 
           st.st_size, st.st_size / (1024.0 * 1024.0));

    // 2. Get allocation size using getxattr
    printf("2. Getting allocation size using getxattr...\n");
    char size_str[64];
    ssize_t size_len = getxattr(path, "user.allocation_size", size_str, sizeof(size_str) - 1);
    if (size_len < 0) {
        print_error("getxattr allocation_size");
        return -1;
    }
    size_str[size_len] = '\0';
    size_t allocation_size = atol(size_str);
    printf("   Child sees allocation size: %s bytes (%.2f MB)\n", 
           size_str, allocation_size / (1024.0 * 1024.0));

    CUmemFabricHandle fabric_handle;
    ssize_t bytes_read = getxattr(path, "user.fabric_handle", &fabric_handle, sizeof(CUmemFabricHandle));
    if (bytes_read != sizeof(CUmemFabricHandle)) {
        printf("getxattr failed: expected %zu bytes, got %zd bytes\n", sizeof(CUmemFabricHandle), bytes_read);
        print_error("getxattr");
        return -1;
    }
    
    printf("4. Successfully read fabric handle (%zu bytes)\n", sizeof(CUmemFabricHandle));
    
    // 5. Initialize CUDA
    CUDA_CHECK_DRV(hipInit(0));
    
    hipDeviceptr_t va = get_va_from_fabric_handle(fabric_handle, allocation_size, allocation_size);

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // 7. Write to the shared memory from child
    kernel_read<<<1, 1, 0, stream>>>((void *)va, allocation_size);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));

    printf("5. Successfully wrote to shared GPU memory from child!\n");
    printf("✅ CHILD PROCESS completed successfully!\n");
    return 0;
}

int main(int argc, char *argv[]) {
    printf("GPU Memory FUSE Filesystem Test Client\n");
    printf("======================================\n");
    
    // Define long options
    static struct option long_options[] = {
        {"parent", no_argument, 0, 'p'},
        {"child",  no_argument, 0, 'c'},
        {"help",   no_argument, 0, 'h'},
        {0, 0, 0, 0}
    };
    
    int option_index = 0;
    int opt;
    enum { MODE_NONE, MODE_PARENT, MODE_CHILD } mode = MODE_NONE;
    
    // Parse command line options
    while ((opt = getopt_long(argc, argv, "pch", long_options, &option_index)) != -1) {
        switch (opt) {
            case 'p':
                mode = MODE_PARENT;
                break;
            case 'c':
                mode = MODE_CHILD;
                break;
            case 'h':
                print_usage(argv[0]);
                return 0;
            default:
                print_usage(argv[0]);
                return 1;
        }
    }
    
    // Check if mode was specified
    if (mode == MODE_NONE) {
        printf("Error: You must specify either --parent or --child\n\n");
        print_usage(argv[0]);
        return 1;
    }
    
    // Check if mount point exists
    struct stat st;
    if (stat(TEST_MOUNT_PATH, &st) != 0) {
        printf("Error: Mount point %s does not exist.\n", TEST_MOUNT_PATH);
        printf("Please make sure the FUSE filesystem is running.\n");
        return 1;
    }

    // Run the appropriate test based on mode
    int result = 0;
    switch (mode) {
        case MODE_PARENT:
            result = test_parent_process();
            break;
        case MODE_CHILD:
            result = test_child_process();
            break;
        default:
            // Should never reach here
            result = 1;
            break;
    }

    return result;
}